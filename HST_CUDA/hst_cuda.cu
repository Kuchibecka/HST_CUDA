#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <fstream>

__global__ void kernel(int* vec, int* mat, int* out, const int N, const int M) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int sum = 0;
	if (tid < M) {
		for (int i = 0; i < N; i++)
			sum += vec[i] * mat[(i * M) + tid];
		out[tid] = sum;
	}
}

void init_array(int* a, const int N);
void init_mat(int* a, const int N, const int M);
void print_array(int* a, const int N, char* d);
void print_mat(int* a, const int N, const int M, char* d);
void generateMatrixInFile(std::string fileName, int fileSize);

using namespace std;

int main(void) {
	int* a, * b, * c;
	int* dev_a, * dev_b, * dev_c;
	int choice1 = 0, choice2 = 0;
	int N, M;

	//std::cin >> choice1;
	//std::cin >> choice2;

	printf("Enter size in Mb if you want to generate a new vector file\n");
	printf("Enter \"0\" if you wanna keep old vector file\n");
	std::cin >> choice1;

	if (choice1) {
		generateMatrixInFile("inV.txt", choice1);
		printf("Generated %d Mb inV.txt file with input vector\n", choice1);
	}

	printf("Enter size in Mb if you want to generate a new matrix file\n");
	printf("Enter \"0\" if you wanna keep old matrix file\n");
	std::cin >> choice2;

	if (choice2) {
		generateMatrixInFile("inM.txt", choice2);
		printf("Generated %d Mb inM.txt file with input matrix\n", choice2);
	}

	printf("Enter vector length (N)\n");
	std::cin >> N;

	printf("Enter matrix size (M)\n");
	std::cin >> M;

	a = (int*)malloc(sizeof(int) * N);
	b = (int*)malloc(sizeof(int) * N * M);
	c = (int*)malloc(sizeof(int) * M);
	printf("Allocated memory for result vector, input vector and matrix\n");

	init_array(a, N);
	printf("Readed input vector from file\n");
	init_mat(b, N, M);
	printf("Readed input matrix from file\n");

	// printf("initial data:\n");
	// print_array(a, N);
	// print_mat(b, N, M, "matrix");

	hipMalloc((void**)&dev_a, sizeof(int) * N);
	hipMalloc((void**)&dev_b, sizeof(int) * N * M);
	hipMalloc((void**)&dev_c, sizeof(int) * M);
	printf("CUDA allocated memory for result vector, matrix and input vector\n");

	hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(int) * N * M, hipMemcpyHostToDevice);
	printf("CUDA copied input matrix and vector\n");

	printf("\n\nRunning kernel with M = %d, N = ...\n\n");

	hipEvent_t start, stop;
	float gpuTime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	kernel << <M / 1024 + 1, 1024 >> > (dev_a, dev_b, dev_c, N, M);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("time on GPU = %.2f ms \n", gpuTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(c, dev_c, sizeof(int) * M, hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	FILE* log;
	log = fopen("log.txt", "a");
	fprintf(log, "Matrix[%d x %d], vector[%d], elapsed time: %.2f ms\n", N, M, N, gpuTime);
	printf("New log appended to log.txt\n");

	FILE* out;
	out = fopen("out.txt", "w");
	for (int i = 0; i < M; i++)
		fprintf(out, "V[%d]: %d\n", i, c[i]);
	printf("Result data written to out.txt\n");

	fclose(log);
	fclose(out);
	free(a);
	free(b);
	free(c);
	//print_array(c, M);

	return 0;
};

void generateMatrixInFile(std::string fileName, int fileSize) {
	srand(time(NULL));
	int bytesFileSize = fileSize * 1024 * 1024;
	int matrixSize = sqrt(bytesFileSize / 4);

	FILE* fd;
	fd = fopen(fileName.c_str(), "w+b");

	for (int i = 0; i < matrixSize; i++) {
		for (int j = 0; j < matrixSize; j++)
			fprintf(fd, "%d ", 1 + rand() % 100);
	}
	fclose(fd);
}

void init_array(int* a, const int N) {
	FILE* inV = fopen("inV.txt", "r");
	if (!inV) {
		printf("Error opening input matrix file");
		exit(1);
	}
	for (int i = 0; i < N; i++) {
		fscanf(inV, "%d", &a[i]);
	}
	fclose(inV);
}

void print_array(int* a, const int N) {
	int i;
	for (i = 0; i < N; i++)
		printf("\nV[%d]: %d", i, a[i]);
	printf("\n");
}

void init_mat(int* a, const int N, const int M) {
	FILE* inM = fopen("inM.txt", "r");
	if (!inM) {
		printf("Error opening input matrix file");
		exit(1);
	}

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			fscanf(inM, "%d", &a[i * M + j]);
		}
	}
	fclose(inM);
}

void print_mat(int* a, const int N, const int M, char* d) {
	int i, j;
	for (i = 0; i < N; i++) {
		printf("\n%s[%d]:", d, i);
		for (j = 0; j < M; j++)
			printf("\t%d", a[i * M + j]);
	}
	printf("\n");
}